#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/all.h>
#include <torch/python.h>

const int BLOCKWIDTH = 256;
const int BLOCKHEIGHT2 = 16;
const int BLOCKHEIGHT3 = 24;
const int BLOCKHEIGHT4 = 32;
const int BLOCKHEIGHT8 = 64;

#define VEC_MATRIX_MATMUL(BIT_NUM, FUN_NAM)                                    \
  void vecquant##BIT_NUM##matmul_cuda(torch::Tensor vec, torch::Tensor mat,    \
                                      torch::Tensor mul, torch::Tensor scales, \
                                      torch::Tensor zeros) {                   \
    int batch = vec.size(0);                                                   \
    int vec_height = vec.size(1);                                              \
    int height = mat.size(0);                                                  \
    int width = mat.size(1);                                                   \
    dim3 blocks((height + BLOCKHEIGHT##BIT_NUM - 1) / BLOCKHEIGHT##BIT_NUM,    \
                (width + BLOCKWIDTH - 1) / BLOCKWIDTH, batch);                 \
    dim3 threads(BLOCKWIDTH);                                                  \
    AT_DISPATCH_FLOATING_TYPES(                                                \
        vec.type(), FUN_NAM, ([&] {                                            \
          VecQuant##BIT_NUM##MatMulKernel<<<blocks, threads>>>(                \
              vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),     \
              scales.data<scalar_t>(), zeros.data<scalar_t>(), batch,          \
              vec_height, height, width);                                      \
        }));                                                                   \
  }

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double *address, double val) {
  unsigned long long int *address_as_ull = (unsigned long long int *)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}
#endif

__device__ inline unsigned int as_unsigned(int i) {
  return *reinterpret_cast<unsigned int *>(&i);
}

template <typename T>
__global__ void
VecQuant2MatMulKernel(const T *__restrict__ vec, const int *__restrict__ mat,
                      T *__restrict__ mul, const T *__restrict__ scales,
                      const T *__restrict__ zeros, int batch, int vec_height,
                      int height, int width) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT2 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ T blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] =
      vec[b * vec_height + (h / BLOCKHEIGHT2) * BLOCKWIDTH + threadIdx.x];
  __syncthreads();

  const T scale = scales[w];
  const T zero = zeros[w];

  T res = 0;
  int i = width * h + w;
  int k = 0;

  unsigned int tmp;

  while (k < BLOCKWIDTH) {
    tmp = as_unsigned(mat[i]);
    res += (scale * T((tmp >> 0) & 0x3) - zero) * blockvec[k + 0];
    res += (scale * T((tmp >> 2) & 0x3) - zero) * blockvec[k + 1];
    res += (scale * T((tmp >> 4) & 0x3) - zero) * blockvec[k + 2];
    res += (scale * T((tmp >> 6) & 0x3) - zero) * blockvec[k + 3];
    res += (scale * T((tmp >> 8) & 0x3) - zero) * blockvec[k + 4];
    res += (scale * T((tmp >> 10) & 0x3) - zero) * blockvec[k + 5];
    res += (scale * T((tmp >> 12) & 0x3) - zero) * blockvec[k + 6];
    res += (scale * T((tmp >> 14) & 0x3) - zero) * blockvec[k + 7];
    res += (scale * T((tmp >> 16) & 0x3) - zero) * blockvec[k + 8];
    res += (scale * T((tmp >> 18) & 0x3) - zero) * blockvec[k + 9];
    res += (scale * T((tmp >> 20) & 0x3) - zero) * blockvec[k + 10];
    res += (scale * T((tmp >> 22) & 0x3) - zero) * blockvec[k + 11];
    res += (scale * T((tmp >> 24) & 0x3) - zero) * blockvec[k + 12];
    res += (scale * T((tmp >> 26) & 0x3) - zero) * blockvec[k + 13];
    res += (scale * T((tmp >> 28) & 0x3) - zero) * blockvec[k + 14];
    res += (scale * T((tmp >> 30) & 0x3) - zero) * blockvec[k + 15];
    i += width;
    k += 16;
  }
  atomicAdd(&mul[b * width + w], res);
}

template <typename T>
__global__ void
VecQuant3MatMulKernel(const T *__restrict__ vec, const int *__restrict__ mat,
                      T *__restrict__ mul, const T *__restrict__ scales,
                      const T *__restrict__ zeros, int batch, int vec_height,
                      int height, int width) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT3 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ T blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] =
      vec[b * vec_height + (h / BLOCKHEIGHT3) * BLOCKWIDTH + threadIdx.x];
  __syncthreads();

  T scale = scales[w];
  T zero = zeros[w];

  T res = 0;
  int i = width * h + w;
  int k = 0;

  unsigned int tmp1;
  unsigned int tmp2;
  unsigned int tmp;

  while (k < BLOCKWIDTH) {
    tmp1 = as_unsigned(mat[i]);
    res += (scale * T((tmp1 >> 0) & 0x7) - zero) * blockvec[k + 0];
    res += (scale * T((tmp1 >> 3) & 0x7) - zero) * blockvec[k + 1];
    res += (scale * T((tmp1 >> 6) & 0x7) - zero) * blockvec[k + 2];
    res += (scale * T((tmp1 >> 9) & 0x7) - zero) * blockvec[k + 3];
    res += (scale * T((tmp1 >> 12) & 0x7) - zero) * blockvec[k + 4];
    res += (scale * T((tmp1 >> 15) & 0x7) - zero) * blockvec[k + 5];
    res += (scale * T((tmp1 >> 18) & 0x7) - zero) * blockvec[k + 6];
    res += (scale * T((tmp1 >> 21) & 0x7) - zero) * blockvec[k + 7];
    res += (scale * T((tmp1 >> 24) & 0x7) - zero) * blockvec[k + 8];
    res += (scale * T((tmp1 >> 27) & 0x7) - zero) * blockvec[k + 9];
    i += width;
    tmp2 = as_unsigned(mat[i]);
    tmp = (tmp1 >> 30) | ((tmp2 << 2) & 0x4);
    tmp2 >>= 1;
    res += (scale * T(tmp) - zero) * blockvec[k + 10];
    k += 11;
    res += (scale * T((tmp2 >> 0) & 0x7) - zero) * blockvec[k + 0];
    res += (scale * T((tmp2 >> 3) & 0x7) - zero) * blockvec[k + 1];
    res += (scale * T((tmp2 >> 6) & 0x7) - zero) * blockvec[k + 2];
    res += (scale * T((tmp2 >> 9) & 0x7) - zero) * blockvec[k + 3];
    res += (scale * T((tmp2 >> 12) & 0x7) - zero) * blockvec[k + 4];
    res += (scale * T((tmp2 >> 15) & 0x7) - zero) * blockvec[k + 5];
    res += (scale * T((tmp2 >> 18) & 0x7) - zero) * blockvec[k + 6];
    res += (scale * T((tmp2 >> 21) & 0x7) - zero) * blockvec[k + 7];
    res += (scale * T((tmp2 >> 24) & 0x7) - zero) * blockvec[k + 8];
    res += (scale * T((tmp2 >> 27) & 0x7) - zero) * blockvec[k + 9];
    i += width;
    tmp1 = as_unsigned(mat[i]);
    tmp = (tmp2 >> 30) | ((tmp1 << 1) & 0x6);
    tmp1 >>= 2;
    res += (scale * T(tmp) - zero) * blockvec[k + 10];
    k += 11;
    res += (scale * T((tmp1 >> 0) & 0x7) - zero) * blockvec[k + 0];
    res += (scale * T((tmp1 >> 3) & 0x7) - zero) * blockvec[k + 1];
    res += (scale * T((tmp1 >> 6) & 0x7) - zero) * blockvec[k + 2];
    res += (scale * T((tmp1 >> 9) & 0x7) - zero) * blockvec[k + 3];
    res += (scale * T((tmp1 >> 12) & 0x7) - zero) * blockvec[k + 4];
    res += (scale * T((tmp1 >> 15) & 0x7) - zero) * blockvec[k + 5];
    res += (scale * T((tmp1 >> 18) & 0x7) - zero) * blockvec[k + 6];
    res += (scale * T((tmp1 >> 21) & 0x7) - zero) * blockvec[k + 7];
    res += (scale * T((tmp1 >> 24) & 0x7) - zero) * blockvec[k + 8];
    res += (scale * T((tmp1 >> 27) & 0x7) - zero) * blockvec[k + 9];
    i += width;
    k += 10;
  }

  atomicAdd(&mul[b * width + w], res);
}

/*
To perform matrix multiplication between a vector and a matrix, where the matrix
elements are quantized to 4 bits. It uses shared memory and atomic operations to
achieve efficient parallel computation on a GPU.

vec: A pointer to the input vector
mat: A pointer to the input matrix
mul: A pointer to the output matrix
scales: A pointer to an array of scale values for each column of the matrix
zeros: A pointer to an array of zero values for each column of the matrix
batch: The number of batches in the input data
vec_height: The height of the input vector
height: The height of the input matrix
width: The width of the input matrix

It uses the __global__ keyword to indicate that it is a CUDA kernel function
that can be launched on a GPU device. The function also uses the __restrict__
keyword to indicate that the input and output pointers do not overlap in memory.
*/
template <typename T>
__global__ void
VecQuant4MatMulKernel(const T *__restrict__ vec, const int *__restrict__ mat,
                      T *__restrict__ mul, const T *__restrict__ scales,
                      const T *__restrict__ zeros, int batch, int vec_height,
                      int height, int width) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT4 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  // Shared memory buffer used to store a portion of the input vector that is
  // used by each thread in the CUDA kernel function
  __shared__ T blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] =
      vec[b * vec_height + (h / BLOCKHEIGHT4) * BLOCKWIDTH + threadIdx.x];
  // By using __syncthreads(), we ensure that all threads have completed loading
  // the blockvec buffer before any thread proceeds to compute the corresponding
  // element of the output matrix. This helps to avoid race conditions and ensure
  // correct results.
  __syncthreads();

  T scale = scales[w];
  T zero = zeros[w];

  T res = 0;
  int i = width * h + w;
  int k = 0;

  unsigned int tmp;

  while (k < BLOCKWIDTH) {
    tmp = as_unsigned(mat[i]);
    res += (scale * T((tmp >> 0) & 0xF) - zero) * blockvec[k + 0];
    res += (scale * T((tmp >> 4) & 0xF) - zero) * blockvec[k + 1];
    res += (scale * T((tmp >> 8) & 0xF) - zero) * blockvec[k + 2];
    res += (scale * T((tmp >> 12) & 0xF) - zero) * blockvec[k + 3];
    res += (scale * T((tmp >> 16) & 0xF) - zero) * blockvec[k + 4];
    res += (scale * T((tmp >> 20) & 0xF) - zero) * blockvec[k + 5];
    res += (scale * T((tmp >> 24) & 0xF) - zero) * blockvec[k + 6];
    res += (scale * T((tmp >> 28) & 0xF) - zero) * blockvec[k + 7];
    i += width;
    k += 8;
  }

  atomicAdd(&mul[b * width + w], res);
}

template <typename T>
__global__ void
VecQuant8MatMulKernel(const T *__restrict__ vec, const int *__restrict__ mat,
                      T *__restrict__ mul, const T *__restrict__ scales,
                      const T *__restrict__ zeros, int batch, int vec_height,
                      int height, int width) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT8 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ T blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] =
      vec[b * vec_height + (h / BLOCKHEIGHT8) * BLOCKWIDTH + threadIdx.x];
  __syncthreads();

  T scale = scales[w];
  T zero = zeros[w];

  T res = 0;
  int i = width * h + w;
  int k = 0;

  unsigned int tmp;

  while (k < BLOCKWIDTH) {
    tmp = as_unsigned(mat[i]);
    res += (scale * T((tmp >> 0) & 0xFF) - zero) * blockvec[k + 0];
    res += (scale * T((tmp >> 8) & 0xFF) - zero) * blockvec[k + 1];
    res += (scale * T((tmp >> 16) & 0xFF) - zero) * blockvec[k + 2];
    res += (scale * T((tmp >> 24) & 0xFF) - zero) * blockvec[k + 3];
    i += width;
    k += 4;
  }

  atomicAdd(&mul[b * width + w], res);
}

VEC_MATRIX_MATMUL(2, "vecquant2matmul_cuda");
VEC_MATRIX_MATMUL(3, "vecquant3matmul_cuda");
VEC_MATRIX_MATMUL(4, "vecquant4matmul_cuda");
VEC_MATRIX_MATMUL(8, "vecquant8matmul_cuda");
