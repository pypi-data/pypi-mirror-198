#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
#include <hip/hip_fp16.h>
#define MIN_VALUE (-1e38)
// typedef at::float fp16;
#define fp16 float
//#define DTYPE __half
#define DTYPE float
__global__ void kernel_wkv_forward(const int B, const int T, const int C,
                               const float *__restrict__ const _w, const float *__restrict__ const _u, const fp16 *__restrict__ const _k, const fp16 *__restrict__ const _v,
                               fp16 *__restrict__ const _y, float *__restrict__ const _aa, float *__restrict__ const _bb, float *__restrict__ const _pp) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;
    const int _state_offset = _b * C + _c;

    float u = _u[_c];
    float w = _w[_c];
    const fp16 *__restrict__ const k = _k + _offset;
    const fp16 *__restrict__ const v = _v + _offset;
    fp16 *__restrict__ const y = _y + _offset;

    float aa = _aa[_state_offset];
    float bb = _bb[_state_offset];
    float pp = _pp[_state_offset];
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        const float kk = float(k[ii]);
        const float vv = float(v[ii]);
        float ww = u + kk;
        float p = max(pp, ww);
        float e1 = exp(pp - p);
        float e2 = exp(ww - p);
        y[ii] = fp16((e1 * aa + e2 * vv) / (e1 * bb + e2));
        ww = w + pp;
        p = max(ww, kk);
        e1 = exp(ww - p);
        e2 = exp(kk - p);
        aa = e1 * aa + e2 * vv;
        bb = e1 * bb + e2;
        pp = p;
    }
    _aa[_state_offset] = aa;
    _bb[_state_offset] = bb;
    _pp[_state_offset] = pp;
}
void cuda_wkv_forward(int B, int T, int C, float *w, float *u, fp16 *k, fp16 *v, fp16 *y, float *aa, float *bb, float *pp) {
    dim3 threadsPerBlock( min(C, 32) );
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_wkv_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, y, aa, bb, pp);
}

DTYPE *cast(fp16 *ptr)
{
    return reinterpret_cast<DTYPE *>(ptr);
}

__global__ void kernel_mm8_seq(
    const int B, const int N, const int M,
    const fp16 *__restrict__ const x, const int x_stride,
    const uint8_t *__restrict__ const w, const int w_stride,
    fp16 *__restrict__ const y, const int y_stride,
    fp16 *__restrict__ const r

    )
{

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int k = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < B && k < M)
    {
        float y_local = 0;
        for (int j = 0; j < N; ++j)
        {
            y_local +=(x[i * x_stride + j]) * (w[k * w_stride + j] * (r[j]));
        }
        y[i * y_stride + k] = (y_local);
    }
}
void cuda_mm8_seq(int B, int N, int M,
                  fp16 *x, int x_stride,
                  uint8_t *w, int w_stride,
                  fp16 *y, int y_stride,
                    fp16 *r
                  )
{
    dim3 blockSize(1, 128);
    dim3 gridSize((B + blockSize.x - 1) / blockSize.x, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm8_seq<<<gridSize, blockSize>>>(
        B, N, M, (x), x_stride, w, w_stride,
        (y), y_stride,(r));
}

#define MM8_ONE_JSPLIT 24
#define MM8_ONE_TILE 1024

__global__ void kernel_mm8_one(
    const int N, const int M,
    const fp16 *__restrict__ const x,
    const uint8_t *__restrict__ const w, const int w_stride,
    fp16 *__restrict__ const y,
    const fp16 *__restrict__ const r
    ){

    const int k = blockIdx.y * blockDim.y + threadIdx.y;
    const int j0 = min(N, blockIdx.x * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));
    const int j1 = min(N, (blockIdx.x + 1) * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));

    if (k < M) {
        fp16 y_local = fp16(0);
        for (int j = j0; j < j1; ++j) {
            y_local += x[j] * (
                (w[j * w_stride + k] * r[j])
                
            );
        }
        atomicAdd(reinterpret_cast<DTYPE *>(&y[k]), *reinterpret_cast<DTYPE *>(&y_local));
    }
}
void cuda_mm8_one(int N, int M,
                  fp16 *x,
                  uint8_t *w, int w_stride,
                  fp16 *y,
                    fp16 *r   
                ) {
    dim3 blockSize(1, MM8_ONE_TILE);
    dim3 gridSize(MM8_ONE_JSPLIT, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm8_one<<<gridSize, blockSize>>>(
        N, M, x, w, w_stride,y, r);
}

__global__ void kernel_mm8_three(
    const int N, const int M,
    const fp16 *__restrict__ const x,
    const fp16 *__restrict__ const x1,
    const fp16 *__restrict__ const x2,
    
    const uint8_t *__restrict__ const w, const int w_stride,
    const uint8_t *__restrict__ const w1, const int w1_stride,
    const uint8_t *__restrict__ const w2, const int w2_stride,
    fp16 *__restrict__ const y,
    fp16 *__restrict__ const y1,
    fp16 *__restrict__ const y2,
    const fp16 *__restrict__ const r,
    const fp16 *__restrict__ const r1,
    const fp16 *__restrict__ const r2
    
    ){

    const int k = blockIdx.y * blockDim.y + threadIdx.y;
    const int j0 = min(N, blockIdx.x * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));
    const int j1 = min(N, (blockIdx.x + 1) * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));

    if (k < M) {
        fp16 y_local = at::Half(0);
        fp16 y1_local = at::Half(0);
        fp16 y2_local = at::Half(0);
        for (int j = j0; j < j1; ++j) {
            y_local += x[j] * (
                (w[j * w_stride + k] * r[j]));
            y1_local += x1[j] * (
                (w1[j * w1_stride + k] * r1[j]));
            y2_local += x2[j] * (
                (w2[j * w2_stride + k] * r2[j]));
           
        }
        atomicAdd(reinterpret_cast<DTYPE *>(&y[k]), *reinterpret_cast<DTYPE *>(&y_local));
        atomicAdd(reinterpret_cast<DTYPE *>(&y1[k]), *reinterpret_cast<DTYPE *>(&y1_local));
        atomicAdd(reinterpret_cast<DTYPE *>(&y2[k]), *reinterpret_cast<DTYPE *>(&y2_local));
    }
}
void cuda_mm8_three(int N, int M,
                    fp16 *x,
                    fp16 *x1,
                    fp16 *x2,
                    uint8_t *w, int w_stride,
                    uint8_t *w1, int w1_stride,
                    uint8_t *w2, int w2_stride,
                    fp16 *y,
                    fp16 *y1,
                    fp16 *y2,
                    fp16 *r  ,
                    fp16 *r1,
                    fp16 *r2 
                ) {
    dim3 blockSize(1, MM8_ONE_TILE);
    dim3 gridSize(MM8_ONE_JSPLIT, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm8_three<<<gridSize, blockSize>>>(
        N, M, x, x1, x2, w, w_stride, w1, w1_stride, w2, w2_stride, y, y1, y2, r, r1, r2);
}

