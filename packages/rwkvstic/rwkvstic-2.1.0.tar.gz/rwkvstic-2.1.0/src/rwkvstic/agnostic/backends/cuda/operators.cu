#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
#include <hip/hip_fp16.h>
#define MIN_VALUE (-1e38)
__global__ void kernel_wkv_forward(const int B, const int T, const int C,
                               const float *__restrict__ const _w, const float *__restrict__ const _u, const float *__restrict__ const _k, const float *__restrict__ const _v,
                               float *__restrict__ const _y, float *__restrict__ const _aa, float *__restrict__ const _bb, float *__restrict__ const _pp) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;
    const int _state_offset = _b * C + _c;

    float u = _u[_c];
    float w = _w[_c];
    const float *__restrict__ const k = _k + _offset;
    const float *__restrict__ const v = _v + _offset;
    float *__restrict__ const y = _y + _offset;

    float aa = _aa[_state_offset];
    float bb = _bb[_state_offset];
    float pp = _pp[_state_offset];
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        // const float kk = exp(k[ii] + u);
        // const float vv = v[ii];
        // const float wr1 = aa + kk * vv;
        // const float wr2 = bb + kk;
        // y[ii] = wr1 / wr2;
        // aa = (exp(w)*aa) + exp(w+k[ii]) * vv;
        // bb = (exp(w)*bb) + exp(w+k[ii]);
        const float kk = float(k[ii]);
        const float vv = float(v[ii]);
        float ww = u + kk;
        float p = max(pp, ww);
        float e1 = exp(pp - p);
        float e2 = exp(ww - p);
        y[ii] = ((e1 * aa + e2 * vv) / (e1 * bb + e2));
        ww = w + pp;
        p = max(ww, kk);
        e1 = exp(ww - p);
        e2 = exp(kk - p);
        aa = e1 * aa + e2 * vv;
        bb = e1 * bb + e2;
        pp = p;

    }
    _aa[_state_offset] = aa;
    _bb[_state_offset] = bb;
    _pp[_state_offset] = pp;
}
__global__ void kernel_wkv_forward(const int B, const int T, const int C,
                               const double *__restrict__ const _w, const double *__restrict__ const _u, const double *__restrict__ const _k, const double *__restrict__ const _v,
                               double *__restrict__ const _y, double *__restrict__ const _aa, double *__restrict__ const _bb, double *__restrict__ const _pp) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;
    const int _state_offset = _b * C + _c;

    double u = _u[_c];
    double w = _w[_c];
    const double *__restrict__ const k = _k + _offset;
    const double *__restrict__ const v = _v + _offset;
    double *__restrict__ const y = _y + _offset;

    double aa = _aa[_state_offset];
    double bb = _bb[_state_offset];
    double pp = _pp[_state_offset];
    for (int i = 0; i < T; i++) {
        const int ii = i * C;

        // const double kk = exp(k[ii] + u);
        // const double vv = v[ii];
        // const double wr1 = aa + kk * vv;
        // const double wr2 = bb + kk;
        // y[ii] = wr1 / wr2;
        // aa = (exp(w)*aa) + exp(w+k[ii]) * vv;
        // bb = (exp(w)*bb) + exp(w+k[ii]);
        const double kk = double(k[ii]);
        const double vv = double(v[ii]);
        double ww = u + kk;
        double p = max(pp, ww);
        double e1 = exp(pp - p);
        double e2 = exp(ww - p);
        y[ii] = ((e1 * aa + e2 * vv) / (e1 * bb + e2));
        ww = w + pp;
        p = max(ww, kk);
        e1 = exp(ww - p);
        e2 = exp(kk - p);
        aa = e1 * aa + e2 * vv;
        bb = e1 * bb + e2;
        pp = p;

    }
    _aa[_state_offset] = aa;
    _bb[_state_offset] = bb;
    _pp[_state_offset] = pp;
}

void cuda_wkv_forward(int B, int T, int C, double *w, double *u, double *k, double *v, double *y, double *aa, double *bb, double *pp) {
    dim3 threadsPerBlock( min(C, 32) );
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_wkv_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, y, aa, bb, pp);
}
void cuda_wkv_forward(int B, int T, int C, float *w, float *u, float *k, float *v, float *y, float *aa, float *bb, float *pp) {
    dim3 threadsPerBlock( min(C, 32) );
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_wkv_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, y, aa, bb, pp);
}



#define MM8_ONE_JSPLIT 128
#define MM8_ONE_TILE 512


__global__ void kernel_mm8_one(
    const int N, const int M,
    const float *__restrict__ const x,
    const uint8_t *__restrict__ const w, const int w_stride,
    float *__restrict__ const y,
    const float *__restrict__ const r
    ){

    const int k = blockIdx.y * blockDim.y + threadIdx.y;
    const int j0 = min(N, blockIdx.x * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));
    const int j1 = min(N, (blockIdx.x + 1) * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));

    if (k < M) {
        float y_local = 0;
        for (int j = j0; j < j1; ++j) {
            y_local += x[j] * (
                (w[j * w_stride + k] * r[j])
                
            );
        }
        atomicAdd(reinterpret_cast<float *>(&y[k]), *reinterpret_cast<float *>(&y_local));
    }
}

__global__ void kernel_mm8_one(
    const int N, const int M,
    const double *__restrict__ const x,
    const uint8_t *__restrict__ const w, const int w_stride,
    double *__restrict__ const y,
    const double *__restrict__ const r
    ){

    const int k = blockIdx.y * blockDim.y + threadIdx.y;
    const int j0 = min(N, blockIdx.x * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));
    const int j1 = min(N, (blockIdx.x + 1) * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));

    if (k < M) {
        double y_local = 0;
        for (int j = j0; j < j1; ++j) {
            y_local += x[j] * (
                (w[j * w_stride + k] * r[j])
                
            );
        }
        atomicAdd(reinterpret_cast<double *>(&y[k]), *reinterpret_cast<double *>(&y_local));
    }
}


__global__ void kernel_mm8_three(
    const int N, const int M,
    const float *__restrict__ const x,
    const float *__restrict__ const x1,
    const float *__restrict__ const x2,
    
    const uint8_t *__restrict__ const w, const int w_stride,
    const uint8_t *__restrict__ const w1, const int w1_stride,
    const uint8_t *__restrict__ const w2, const int w2_stride,
    float *__restrict__ const y,
    float *__restrict__ const y1,
    float *__restrict__ const y2,
    const float *__restrict__ const r,
    const float *__restrict__ const r1,
    const float *__restrict__ const r2
    
    ){

    const int k = blockIdx.y * blockDim.y + threadIdx.y;
    const int j0 = min(N, blockIdx.x * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));
    const int j1 = min(N, (blockIdx.x + 1) * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));

    if (k < M) {
        float y_local = float(0);
        float y1_local =float(0);
        float y2_local = float(0);
        for (int j = j0; j < j1; ++j) {
            y_local += x[j] * (
                (w[j * w_stride + k] * r[j]));
            y1_local += x1[j] * (
                (w1[j * w1_stride + k] * r1[j]));
            y2_local += x2[j] * (
                (w2[j * w2_stride + k] * r2[j]));
           
        }
        atomicAdd(reinterpret_cast<float *>(&y[k]), *reinterpret_cast<float *>(&y_local));
        atomicAdd(reinterpret_cast<float *>(&y1[k]), *reinterpret_cast<float *>(&y1_local));
        atomicAdd(reinterpret_cast<float *>(&y2[k]), *reinterpret_cast<float *>(&y2_local));
    }
}

__global__ void kernel_mm8_three(
    const int N, const int M,
    const double *__restrict__ const x,
    const double *__restrict__ const x1,
    const double *__restrict__ const x2,

    const uint8_t *__restrict__ const w, const int w_stride,
    const uint8_t *__restrict__ const w1, const int w1_stride,
    const uint8_t *__restrict__ const w2, const int w2_stride,
    double *__restrict__ const y,
    double *__restrict__ const y1,
    double *__restrict__ const y2,
    const double *__restrict__ const r,
    const double *__restrict__ const r1,
    const double *__restrict__ const r2
    
    ){

    const int k = blockIdx.y * blockDim.y + threadIdx.y;
    const int j0 = min(N, blockIdx.x * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));
    const int j1 = min(N, (blockIdx.x + 1) * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));

    if (k < M) {
        double y_local = 0;
        double y1_local = 0;
        double y2_local = 0;
        for (int j = j0; j < j1; ++j) {
            y_local += x[j] * (
                (w[j * w_stride + k] * r[j]));
            y1_local += x1[j] * (
                (w1[j * w1_stride + k] * r1[j]));
            y2_local += x2[j] * (
                (w2[j * w2_stride + k] * r2[j]));
           
        }
        atomicAdd(reinterpret_cast<double *>(&y[k]), *reinterpret_cast<double *>(&y_local));
        atomicAdd(reinterpret_cast<double *>(&y1[k]), *reinterpret_cast<double *>(&y1_local));
        atomicAdd(reinterpret_cast<double *>(&y2[k]), *reinterpret_cast<double *>(&y2_local));
    }
}
// generic T either float or fp16 or fp64

void cuda_mm8_three(int N, int M,
                    double *x,
                    double *x1,
                    double *x2,
                    uint8_t *w, int w_stride,
                    uint8_t *w1, int w1_stride,
                    uint8_t *w2, int w2_stride,
                    double *y,
                    double *y1,
                    double *y2,
                    double *r  ,
                    double *r1,
                    double *r2 
                ) {
    dim3 blockSize(1, MM8_ONE_TILE);
    dim3 gridSize(MM8_ONE_JSPLIT, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm8_three<<<gridSize, blockSize>>>(
        N, M, x, x1, x2, w, w_stride, w1, w1_stride, w2, w2_stride, y, y1, y2, r, r1, r2);
}
void cuda_mm8_one(int N, int M,
                  double *x,
                  uint8_t *w, int w_stride,
                  double *y,
                    double *r   
                ) {
    dim3 blockSize(1, MM8_ONE_TILE);
    dim3 gridSize(MM8_ONE_JSPLIT, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm8_one<<<gridSize, blockSize>>>(
        N, M, x, w, w_stride,y, r);
}
void cuda_mm8_three(int N, int M,
                    float *x,
                    float *x1,
                    float *x2,
                    uint8_t *w, int w_stride,
                    uint8_t *w1, int w1_stride,
                    uint8_t *w2, int w2_stride,
                    float *y,
                    float *y1,
                    float *y2,
                    float *r  ,
                    float *r1,
                    float *r2 
                ) {
    dim3 blockSize(1, MM8_ONE_TILE);
    dim3 gridSize(MM8_ONE_JSPLIT, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm8_three<<<gridSize, blockSize>>>(
        N, M, x, x1, x2, w, w_stride, w1, w1_stride, w2, w2_stride, y, y1, y2, r, r1, r2);
}
void cuda_mm8_one(int N, int M,
                  float *x,
                  uint8_t *w, int w_stride,
                  float *y,
                    float *r   
                ) {
    dim3 blockSize(1, MM8_ONE_TILE);
    dim3 gridSize(MM8_ONE_JSPLIT, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm8_one<<<gridSize, blockSize>>>(
        N, M, x, w, w_stride,y, r);
}
